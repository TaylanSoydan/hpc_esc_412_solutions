#include "hip/hip_runtime.h"
#include <blitz/array.h>
#include <fftw3.h>
#include <complex>
#include <cmath>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftw.h>

using namespace blitz;
using std::complex;

void fill_array(Array<float, 2> &data) {
    // Set the grid to the sum of two sine functions
    for (int i = 0; i < data.rows(); i++) {
        for (int j = 0; j < data.cols(); j++) {
            float x = (float)i / 25.0; // Period of 1/4 of the box in x
            float y = (float)j / 10.0; // Period of 1/10 of the box in y
            data(i, j) = sin(2.0 * M_PI * x) + sin(2.0 * M_PI * y);
        }
    }
}

// Verify the FFT (kdata) of data by performing a reverse transform and comparing
bool validate(Array<float, 2> &data, Array<std::complex<float>, 2> kdata) {
    Array<float, 2> rdata(data.extent());
    fftwf_plan plan = fftwf_plan_dft_c2r_2d(data.rows(), data.cols(),
    //               reinterpret_cast<fftwf_complex *>(kdata.data()), rdata.data(), FFTW_ESTIMATE);
    fftwf_execute(plan);
    fftwf_destroy_plan(plan);
    rdata /= data.size(); // Normalize for the FFT
    return all(abs(data - rdata) < 1e-5);
}
    // Ex1
    //int main() {
    //    int n = 10000;

    //    // Out of place
    //    Array<float, 2> rdata1(n, n);
    //    Array<std::complex<float>, 2> kdata1(n, n / 2 + 1);

    //    fftwf_plan plan1 = fftwf_plan_dft_r2c_2d(n, n,
    //        rdata1.data(), reinterpret_cast<fftwf_complex *>(kdata1.data()), FFTW_ESTIMATE);

    //    fill_array(rdata1);
    //    fftwf_execute(plan1);
    //    fftwf_destroy_plan(plan1);

    //    std::cout << ">>> Out of place FFT " << (validate(rdata1, kdata1) ? "match" : "MISMATCH") << std::endl;

    //    // In-place
    //    Array<float, 2> raw_data2(n, n + 2);
    //    Array<float, 2> rdata2 = raw_data2(Range(0, n - 1), Range(0, n - 1));

    //    fftwf_plan plan2 = fftwf_plan_dft_r2c_2d(n, n,
    //        rdata2.data(), reinterpret_cast<fftwf_complex *>(rdata2.data()), FFTW_ESTIMATE);

    //    fill_array(rdata2);
    //    fftwf_execute(plan2);
    //    fftwf_destroy_plan(plan2);

    //    Array<std::complex<float>, 2> kdata2(reinterpret_cast<std::complex<float> *>(rdata2.data()),
    //        shape(n, n / 2 + 1), neverDeleteData);

    //    std::cout << ">>> In-place FFT " << (validate(rdata1, kdata2) ? "match" : "MISMATCH") << std::endl;

    //    // Transfer data3 to GPU and perform in-place FFT
    //    Array<float, 2> raw_data3(n, n + 2);
    //    Array<float, 2> data3 = raw_data3(blitz::Range::all(), blitz::Range(0, n - 1));

    //    fill_array(data3);

    //    size_t size_in_bytes = sizeof(float) * n * (n + 2);

    //    // Allocate memory on the GPU
    //    void *device_data;
    //    hipMalloc(&device_data, size_in_bytes);

    //    // Copy data from CPU to GPU
    //    hipMemcpy(device_data, data3.data(), size_in_bytes, hipMemcpyHostToDevice);
    //    hipDeviceSynchronize();

    //    hipMemcpy(raw_data4.data(), device_data, size_in_bytes, hipMemcpyDeviceToHost);
    //    std::cout << (validate2d(raw_data3, raw_data4) ? "passed" : "failed") << std::endl;

    //    hipFree(device_data);

    //    // Perform in-place FFT on GPU
    //    Array<float, 2> raw_data5(n, n + 2);
    //    Array<float, 2> data5 = raw_data5(blitz::Range::all(), blitz::Range(0, n - 1));
    //    Array<std::complex<float>, 2> data6(n, n / 2 + 1);

    //    float *device_data;
    //    size_t size_in_byte = sizeof(float) * n * (n + 2);
    //    hipMalloc(&device_data, size_in_byte);
    //    hipMemcpy(device_data, raw_data5.data(), size_in_byte, hipMemcpyHostToDevice);
    //    hipDeviceSynchronize();

    //    int dims[2] = { n, n };
    //    int inembed[2] = { n, n + 2 };
    //    int onembed[2] = { n, n / 2 + 1 };
    //    int batch = 1;
    //    int odist = 1;
    //    int idist = 2;
    //    int istride = 1;
    //    int ostride = 1;

    //    hipfftHandle plan;
    //    hipfftCreate(&plan);
    //    hipfftSetAutoAllocation(plan, HIPFFT_NO_WORKSPACE);
    //    hipfftMakePlanMany(plan, 2, dims, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, batch);

    //    size_t workSize;
    //    hipfftGetSize(plan, &workSize);

    //    void *workArea;
    //    hipMalloc(&workArea, workSize);

    //    hipfftSetWorkArea(plan, workArea);

    //    hipfftExecR2C(plan, reinterpret_cast<hipfftReal *>(device_data),
    //        reinterpret_cast<hipfftComplex *>(device_data));

    //    hipDeviceSynchronize();
    //    hipfftDestroy(plan);

    //    // Copy data from GPU back to CPU
    //    hipMemcpy(data6.data(), device_data, size_in_bytes, hipMemcpyDeviceToHost);

    //    // Synchronize GPU
    //    hipDeviceSynchronize();

    //    // Compare data3 and data4
    //    std::cout << ">>> Data Comparison " << (validate(data5, data6) ? "match" : "MISMATCH") << std::endl;

    //    // Free GPU memory
    //    hipFree(device_data);
    //    hipFree(workArea);

    //    return 0;
    //}
    // EX 2
    //int main() {
    //    int n = 10000;

    //    // Create a new stream
    //    hipStream_t stream;
    //    hipStreamCreate(&stream);

    //    // Out of place
    //    Array<float, 2> rdata1(n, n);
    //    Array<std::complex<float>, 2> kdata1(n, n / 2 + 1);

    //    fftwf_plan plan1 = fftwf_plan_dft_r2c_2d(n, n,
    //        rdata1.data(), reinterpret_cast<fftwf_complex *>(kdata1.data()), FFTW_ESTIMATE);

    //    fill_array(rdata1);
    //    fftwf_execute(plan1);
    //    fftwf_destroy_plan(plan1);

    //    std::cout << ">>> Out of place FFT " << (validate(rdata1, kdata1) ? "match" : "MISMATCH") << std::endl;

    //    // In-place
    //    Array<float, 2> raw_data2(n, n + 2);
    //    Array<float, 2> rdata2 = raw_data2(Range(0, n - 1), Range(0, n - 1));

    //    fftwf_plan plan2 = fftwf_plan_dft_r2c_2d(n, n,
    //        rdata2.data(), reinterpret_cast<fftwf_complex *>(rdata2.data()), FFTW_ESTIMATE);

    //    fill_array(rdata2);
    //    fftwf_execute(plan2);
    //    fftwf_destroy_plan(plan2);

    //    Array<std::complex<float>, 2> kdata2(reinterpret_cast<std::complex<float> *>(rdata2.data()),
    //        shape(n, n / 2 + 1), neverDeleteData);

    //    std::cout << ">>> In-place FFT " << (validate(rdata1, kdata2) ? "match" : "MISMATCH") << std::endl;

    //    // Transfer data3 to GPU and perform in-place FFT
    //    Array<float, 2> raw_data3(n, n + 2);
    //    Array<float, 2> data3 = raw_data3(blitz::Range::all(), blitz::Range(0, n - 1));

    //    fill_array(data3);

    //    size_t size_in_bytes = sizeof(float) * n * (n + 2);

    //    // Allocate memory on the GPU with the new stream
    //    void *device_data;
    //    hipMallocAsync(&device_data, size_in_bytes, stream);

    //    // Copy data from CPU to GPU with the new stream
    //    hipMemcpyAsync(device_data, data3.data(), size_in_bytes, hipMemcpyHostToDevice, stream);

    //    // Synchronize the new stream
    //    hipStreamSynchronize(stream);

    //    hipMemcpy(raw_data4.data(), device_data, size_in_bytes, hipMemcpyDeviceToHost);
    //    std::cout << (validate2d(raw_data3, raw_data4) ? "passed" : "failed") << std::endl;

    //    hipFree(device_data);

    //    // Perform in-place FFT on GPU
    //    Array<float, 2> raw_data5(n, n + 2);
    //    Array<float, 2> data5 = raw_data5(blitz::Range::all(), blitz::Range(0, n - 1));
    //    Array<std::complex<float>, 2> data6(n, n / 2 + 1);

    //    float *device_data;
    //    size_t size_in_byte = sizeof(float) * n * (n + 2);
    //    hipMallocAsync(&device_data, size_in_byte, stream);
    //    hipMemcpyAsync(device_data, raw_data5.data(), size_in_byte, hipMemcpyHostToDevice, stream);

    //    // Create cuFFT plan
    //    int dims[2] = { n, n };
    //    int inembed[2] = { n, n + 2 };
    //    int onembed[2] = { n, n / 2 + 1 };
    //    int batch = 1;
    //    int odist = 1;
    //    int idist = 2;
    //    int istride = 1;
    //    int ostride = 1;

    //    hipfftHandle plan;
    //    hipfftCreate(&plan);
    //    hipfftSetAutoAllocation(plan, HIPFFT_NO_WORKSPACE);
    //    hipfftMakePlanMany(plan, 2, dims, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, batch);

    //    // Set the stream and work area
    //    hipfftSetStream(plan, stream);
    //    hipfftSetWorkArea(plan, device_data);

    //    // Execute the cuFFT plan
    //    hipfftExecR2C(plan, reinterpret_cast<hipfftReal *>(device_data),
    //        reinterpret_cast<hipfftComplex *>(device_data));

    //    // Synchronize the stream
    //    hipStreamSynchronize(stream);

    //    hipfftDestroy(plan);

    //    // Copy data from GPU back to CPU with the new stream
    //    hipMemcpyAsync(data6.data(), device_data, size_in_bytes, hipMemcpyDeviceToHost, stream);

    //    // Synchronize the new stream
    //    hipStreamSynchronize(stream);

    //    // Compare data3 and data4
    //    std::cout << ">>> Data Comparison " << (validate(data5, data6) ? "match" : "MISMATCH") << std::endl;

    //    // Free GPU memory
    //    hipFree(device_data);

    //    // Destroy the stream
    //    hipStreamDestroy(stream);

    //    return 0;
    //}

 // Ex3
int main() {
    int n = 10000;
    int n_streams = 3;

    // Create an array of streams
    hipStream_t streams[n_streams];
    for (int i = 0; i < n_streams; ++i) {
        hipStreamCreate(&streams[i]);
    }

    // Create arrays for GPU slabs and work areas for each stream
    void *gpu_slabs[n_streams];
    void *gpu_works[n_streams];
    size_t slab_size = sizeof(float) * n * (n + 2);
    size_t work_size = calculate_work_size(n);

    for (int i = 0; i < n_streams; ++i) {
        hipMallocAsync(&gpu_slabs[i], slab_size, streams[i]);
        hipMallocAsync(&gpu_works[i], work_size, streams[i]);
    }

    // Out of place
    Array<float, 2> rdata1(n, n);
    Array<std::complex<float>, 2> kdata1(n, n / 2 + 1);
    fftwf_plan plan1 = fftwf_plan_dft_r2c_2d(n, n,
        rdata1.data(), reinterpret_cast<fftwf_complex *>(kdata1.data()), FFTW_ESTIMATE);
    fill_array(rdata1);

    // Process each slab
    int stream_index = 0;
    for (int i = 0; i < n; ++i) {
        hipMemcpyAsync(gpu_slabs[stream_index], rdata1.data(), slab_size, hipMemcpyHostToDevice, streams[stream_index]);
        hipfftSetStream(plan1, streams[stream_index]);
        hipfftSetWorkArea(plan1, gpu_works[stream_index]);
        hipfftExecR2C(plan1, reinterpret_cast<hipfftReal *>(gpu_slabs[stream_index]),
            reinterpret_cast<hipfftComplex *>(gpu_slabs[stream_index]));
        hipMemcpyAsync(rdata1.data(), gpu_slabs[stream_index], slab_size, hipMemcpyDeviceToHost, streams[stream_index]);
        stream_index = (stream_index + 1) % n_streams;
    }

    fftwf_destroy_plan(plan1);

    std::cout << ">>> Out of place FFT " << (validate(rdata1, kdata1) ? "match" : "MISMATCH") << std::endl;

    // Free GPU memory
    for (int i = 0; i < n_streams; ++i) {
        hipFree(gpu_slabs[i]);
        hipFree(gpu_works[i]);
    }

    // Destroy streams
    for (int i = 0; i < n_streams; ++i) {
        hipStreamDestroy(streams[i]);
    }

    return 0;
}
